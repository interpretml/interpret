#include "hip/hip_runtime.h"
﻿// Copyright (c) 2018 Microsoft Corporation
// Licensed under the MIT license.
// Author: Paul Koch <code@koch.ninja>

#include "hip/hip_runtime.h"
#include ""

#include <type_traits>

#include "ebm_native.h"
#include "logging.h"
#include "common_c.h"
#include "bridge_c.h"
#include "zones.h"

#include "common_cpp.hpp"
#include "bridge_cpp.hpp"

#include "Registration.hpp"
#include "Loss.hpp"

namespace DEFINED_ZONE_NAME {
#ifndef DEFINED_ZONE_NAME
#error DEFINED_ZONE_NAME must be defined
#endif // DEFINED_ZONE_NAME

template <typename TLoss>
GPU_GLOBAL void TestGpuAdd(const Loss * const pLoss, const int * const pVal1, const int * const pVal2, int * const pResult) {
   TLoss * const pLossSpecific = static_cast<TLoss *>(pLoss);
   const size_t iGpuThread = threadIdx.x;
   pResult[iGpuThread] = static_cast<int>(static_cast<float>(pLossSpecific->CalculateGradient(static_cast<float>(pVal1[iGpuThread]), static_cast<float>(pVal2[iGpuThread]))));
}

struct Cuda_32_Operators final {
   // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html#group__CUDA__MATH__SINGLE
   // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__DOUBLE.html#group__CUDA__MATH__DOUBLE

   static constexpr size_t countPackedItems = 1; // the number of Unpacked items in a Packed structure
   typedef float Unpacked;
   typedef float Packed;

private:

   Packed m_data;

public:

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators() noexcept {
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators(const float data) noexcept : m_data(static_cast<Unpacked>(data)) {
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators(const double data) noexcept : m_data(static_cast<Unpacked>(data)) {
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators(const int data) noexcept : m_data(static_cast<Unpacked>(data)) {
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators operator+ (const Cuda_32_Operators & other) const noexcept {
      return Cuda_32_Operators(m_data + other.m_data);
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators operator- (const Cuda_32_Operators & other) const noexcept {
      return Cuda_32_Operators(m_data - other.m_data);
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators operator* (const Cuda_32_Operators & other) const noexcept {
      return Cuda_32_Operators(m_data * other.m_data);
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators operator/ (const Cuda_32_Operators & other) const noexcept {
      return Cuda_32_Operators(m_data / other.m_data);
   }

   GPU_BOTH INLINE_ALWAYS bool IsAnyEqual(const Cuda_32_Operators & other) const noexcept {
      return m_data == other.m_data;
   }

   GPU_BOTH INLINE_ALWAYS operator float() const noexcept {
      return m_data;
   }

   GPU_BOTH INLINE_ALWAYS operator double() const noexcept {
      return m_data;
   }

   GPU_BOTH INLINE_ALWAYS bool IsAnyInf() const noexcept {
      return isinf(m_data);
   }

   GPU_BOTH INLINE_ALWAYS bool IsAnyNaN() const noexcept {
      return isnan(m_data);
   }

   GPU_BOTH INLINE_ALWAYS Cuda_32_Operators Sqrt() const noexcept {
      return Cuda_32_Operators(sqrtf(m_data));
   }

   template<template <typename, typename, ptrdiff_t, ptrdiff_t, bool> class TExecute, typename TLoss, typename TFloat, ptrdiff_t cCompilerScores, ptrdiff_t cCompilerPack, bool bHessian>
   INLINE_RELEASE_TEMPLATED static ErrorEbm ApplyTraining(const Loss * const pLoss, ApplyTrainingData * const pData) noexcept {
      static constexpr size_t k_cItems = 5;

      bool bExitError = true;

      const int aVal1[k_cItems] = { 5, 4, 3, 2, 1 };
      const int aVal2[k_cItems] = { 100, 200, 300, 400, 500 };
      int aResult[k_cItems];

      static_assert(std::is_standard_layout<TLoss>::value &&
         std::is_trivially_copyable<TLoss>::value,
         "This allows offsetof, memcpy, memset, inter-language, GPU and cross-machine use where needed");

      int * aDeviceVal1 = nullptr;
      int * aDeviceVal2 = nullptr;
      int * aDeviceResult = nullptr;
      void * pDeviceLoss = nullptr;
      hipError_t error;

      error = hipSetDevice(0);
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMalloc((void **)&aDeviceVal1, k_cItems * sizeof(int));
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMalloc((void **)&aDeviceVal2, k_cItems * sizeof(int));
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMalloc((void **)&aDeviceResult, k_cItems * sizeof(int));
      if(hipSuccess != error) {
         goto exit_error;
      }

      if(!std::is_empty<TLoss>::value) {
         error = hipMalloc((void **)&pDeviceLoss, sizeof(TLoss));
         if(hipSuccess != error) {
            goto exit_error;
         }
         error = hipMemcpy(pDeviceLoss, pLoss, sizeof(TLoss), hipMemcpyHostToDevice);
         if(hipSuccess != error) {
            goto exit_error;
         }
      }

      error = hipMemcpy(aDeviceVal1, aVal1, k_cItems * sizeof(int), hipMemcpyHostToDevice);
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMemcpy(aDeviceVal2, aVal2, k_cItems * sizeof(int), hipMemcpyHostToDevice);
      if(hipSuccess != error) {
         goto exit_error;
      }

      TestGpuAdd<TLoss><<<1, k_cItems>>>(static_cast<Loss *>(pDeviceLoss), aDeviceVal1, aDeviceVal2, aDeviceResult);
      ExecuteApplyTraining<TExecute, TLoss, TFloat, cCompilerScores, cCompilerPack, bHessian><<<1, k_cItems>>>(
         pLoss,
         pData->m_cRuntimeScores,
         pData->m_cRuntimePack
      );

      error = hipGetLastError();
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipDeviceSynchronize();
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMemcpy(aResult, aDeviceResult, k_cItems * sizeof(int), hipMemcpyDeviceToHost);
      if(hipSuccess != error) {
         goto exit_error;
      }

      bExitError = false;

   exit_error:

      bool bExitHard = false;

      if(nullptr != pDeviceLoss) {
         error = hipFree(pDeviceLoss);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(nullptr != aDeviceResult) {
         error = hipFree(aDeviceResult);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(nullptr != aDeviceVal2) {
         error = hipFree(aDeviceVal2);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(nullptr != aDeviceVal1) {
         error = hipFree(aDeviceVal1);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(bExitHard) {
         bExitError = true;

         // not much to do with the error if we fail hipDeviceReset after failing hipFree
         error = hipDeviceReset();
      }

      return bExitError ? Error_UnexpectedInternal : Error_None;
   }

   template<template <typename, typename, ptrdiff_t, ptrdiff_t, bool> class TExecute, typename TLoss, typename TFloat, ptrdiff_t cCompilerScores, ptrdiff_t cCompilerPack, bool bHessian>
   INLINE_RELEASE_TEMPLATED static ErrorEbm ApplyValidation(const Loss * const pLoss, ApplyValidationData * const pData) noexcept {
      // this allows us to switch execution onto GPU, FPGA, or other local computation

      // TODO: use something other than <<<1, 1>>>
      ExecuteApplyValidation<TExecute, TLoss, TFloat, cCompilerScores, cCompilerPack, bHessian><<<1, 1>>>(
         pLoss,
         pData->m_cRuntimeScores,
         pData->m_cRuntimePack,
         nullptr
      );
      return Error_None;
   }
};
static_assert(std::is_standard_layout<Cuda_32_Operators>::value &&
   std::is_trivially_copyable<Cuda_32_Operators>::value,
   "This allows offsetof, memcpy, memset, inter-language, GPU and cross-machine use where needed");

// FIRST, define the RegisterLoss function that we'll be calling from our registrations.  This is a static 
// function, so we can have duplicate named functions in other files and they'll refer to different functions
template<template <typename> class TRegistrable, typename... Args>
INLINE_ALWAYS static std::shared_ptr<const Registration> RegisterLoss(const char * const sRegistrationName, const Args...args) {
   return Register<TRegistrable, Cuda_32_Operators>(sRegistrationName, args...);
}

// now include all our special loss registrations which will use the RegisterLoss function we defined above!
#include "loss_registrations.hpp"

INTERNAL_IMPORT_EXPORT_BODY ErrorEbm CreateLoss_Cuda_32(
   const Config * const pConfig,
   const char * const sLoss,
   const char * const sLossEnd,
   LossWrapper * const pLossWrapperOut
) {
   return Loss::CreateLoss(&RegisterLosses, pConfig, sLoss, sLossEnd, pLossWrapperOut);
}

} // DEFINED_ZONE_NAME
